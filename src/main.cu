#include "hip/hip_runtime.h"
#include <iostream>
#include "VoxelHash.h"
#include <hip/hip_runtime.h>
#include <>

// TODO: need to test insert and query. For now, just test the hash computation.
// TODO: need to fill d_count by calling API.
__global__ void test(HashData* hash, int* d_count) {
    float a = 56 * blockIdx.x;
    float b = 56 * blockIdx.x;
    float c = 56 * blockIdx.x;

    float3 worldpos = make_float3(a, b, c);
    uint pos = hash->computeHashPos(worldpos);

    // output the postion of the voxel on the GPU.
    printf("voxel pos: %d\n", pos);

    return;
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}

int main() {
    HashData hash;
    hash.allocate(true); // inside allocate, initializeHashParams is called. and the c_hashParams is initialized using config file.

    HashData* d_hashdata; // used to get hash data's GPU memory address.

    int count = 0; // on CPU, initialized to 0 for there are no occupied blocks at the beginning.
    int* d_count; // used to get count's GPU memory address for debug insert and query. Not used for now. TODO: need to fill d_count by calling API.

    // allocate memory for d_hashdata and d_count.
    checkCudaError(hipMalloc(&d_hashdata, sizeof(HashData)));
    checkCudaError(hipMalloc(&d_count, sizeof(int)));

    // copy hash data and count to GPU.
    checkCudaError(hipMemcpy(d_hashdata, &hash, sizeof(HashData), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice));

    // launch test kernel.
    dim3 blockSize(1, 1);
    dim3 gridSize(1000);
    test<<<gridSize, blockSize>>>(d_hashdata, d_count);
    checkCudaError(hipDeviceSynchronize());

    // free memory.
    hash.free();

    // copy count back to CPU for debug and result check.
    checkCudaError(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "count: " << count << std::endl;

    // free GPU memory.
    checkCudaError(hipFree(d_hashdata));
    checkCudaError(hipFree(d_count));

    return 0;
}
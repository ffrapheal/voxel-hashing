#include "hip/hip_runtime.h"
#include <iostream>
#include "VoxelHash.h"
#include <hip/hip_runtime.h>
#include <>
#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>
__global__ void test(HashData * hash,int * count,float * pos,int * num_points)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=*num_points) return;
    float3 worldpos=make_float3(pos[idx*3],pos[idx*3+1],pos[idx*3+2]);
    //assume (a b c) is the coordinate of a point.
    bool insert;
    int3 voxelpos = hash->worldToVirtualVoxelPos(worldpos);
    uint hashpos=hash->computeHashPos(worldpos);
    insert=hash->insertHashEntryElement(worldpos);
    __threadfence();
    HashEntry curr = hash->getHashEntry(worldpos);
    uint h = hash->computeHashPos(worldpos);
    if(curr.ptr!=FREE_ENTRY)
    {
        atomicAdd(&count[0], 1);
        Voxel v = hash->getVoxel(worldpos);
    }
    //printf("a:%d b:%d c:%d pos:%d ptr:%d insert:%s\n",voxelpos.x,voxelpos.y,voxelpos.z,pos,curr.ptr,insert ? "true" : "false");
    return;
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}

int main() {
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud(new pcl::PointCloud<pcl::PointXYZ>);

    // 读取PCD文件
    if (pcl::io::loadPCDFile<pcl::PointXYZ>("/home/zzz/code/hash/point_cloud_7377_points.pcd", *cloud) == -1) // 这里替换为你的PCD文件路径
    {
        PCL_ERROR("无法读取文件 \n");
        return -1;
    }
    // 输出点云信息
    std::cout << "点云宽度: " << cloud->width << std::endl;
    std::cout << "点云高度: " << cloud->height << std::endl;
    std::cout << "点云大小: " << cloud->points.size() << std::endl;
    // 遍历点云中的每个点
    size_t num_points = cloud->points.size();
    float* host_points = new float[num_points * 3]; // 每个点有三个坐标
    for (size_t i = 0; i < num_points; ++i) {
        host_points[i * 3] = cloud->points[i].x;
        host_points[i * 3 + 1] = cloud->points[i].y;
        host_points[i * 3 + 2] = cloud->points[i].z;
    }

    // 分配GPU内存
    float* device_points;
    hipMalloc(&device_points, num_points * 3 * sizeof(float));
    int* device_num_points;
    hipMalloc(&device_num_points,sizeof(int));
    // 复制数据到GPU
    hipMemcpy(device_points, host_points, num_points * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_num_points, &num_points, sizeof(int), hipMemcpyHostToDevice);
    delete[] host_points;
    HashData hash;
    HashParams params;
    hash.allocate(true);
    HashData *d_hashdata;
    int count=0;
    int *d_count;
    hipMalloc(&d_hashdata,sizeof(HashData));
    hipMalloc(&d_count,sizeof(int));
    hipMemcpy(d_hashdata,&hash,sizeof(HashData),hipMemcpyHostToDevice);
    hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
    dim3 blockSize(1024);
    dim3 gridSize((num_points + blockSize.x - 1) / blockSize.x);
    test<<<gridSize,blockSize>>>(d_hashdata,d_count,device_points,device_num_points);
    hipDeviceSynchronize();
    hash.free();
    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);    
    std::cout<<"count: "<<count<<std::endl;
    hipFree(d_hashdata);
    hipFree(device_points);
    hipFree(d_count);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after kernel launch: %s\n", hipGetErrorString(err));
    }
    return 0;
}
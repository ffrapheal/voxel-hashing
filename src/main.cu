#include "hip/hip_runtime.h"
#include <iostream>
#include "VoxelHash.h"
#include <hip/hip_runtime.h>
#include <>
#include <pcl/point_types.h>
__global__ void test(HashData * hash,int * count,float * pos,int num_points)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=num_points) return;
    float3 worldpos=make_float3(pos[idx*3],pos[idx*3+1],pos[idx*3+2]);
    //assume (a b c) is the coordinate of a point.
    int3 voxelpos = hash->worldToVirtualVoxelPos(worldpos);
    uint hashpos=hash->computeHashPos(worldpos);
    hash->insertHashEntryElement(worldpos);
    __threadfence();
    HashEntry curr = hash->getHashEntryForWorldPos(worldpos);
    uint h = hash->computeHashPos(worldpos);
    if(curr.ptr!=FREE_ENTRY)
    {
        atomicAdd(&count[0], 1);
        Voxel * v = hash->getVoxel(worldpos);
        printf("voxelpos: %d %d %d\n",voxelpos.x,voxelpos.y,voxelpos.z);
        printf("voxel: %f %f %f\n",v->sdf_sum,v->weight_sum,v->sdf_sum/v->weight_sum);
    }
    return;
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}

int main() {
    pcl::PointCloud<pcl::PointXYZINormal>::Ptr cloud(new pcl::PointCloud<pcl::PointXYZINormal>);
    printf("hello world\n");
    // read pcd file
    if (pcl::io::loadPCDFile<pcl::PointXYZINormal>("/home/hmy/voxel_hashing_dev/point_cloud_7377_points.pcd", *cloud) == -1) // 这里替换为你的PCD文件路径
    {
        PCL_ERROR("failed to read pcd file \n");
        return -1;
    }
    // print point cloud info
    std::cout << "point cloud width: " << cloud->width << std::endl;
    std::cout << "point cloud height: " << cloud->height << std::endl;
    std::cout << "point cloud size: " << cloud->points.size() << std::endl;
    // traverse each point in point cloud
    size_t num_points = cloud->points.size();
    float* host_points = new float[num_points * 3]; // each point has three coordinates
    for (size_t i = 0; i < num_points; ++i) {
        host_points[i * 3] = cloud->points[i].x;
        host_points[i * 3 + 1] = cloud->points[i].y;
        host_points[i * 3 + 2] = cloud->points[i].z;
    }

    // allocate memory on gpu
    float* device_points;
    hipMalloc(&device_points, num_points * 3 * sizeof(float));
    int* device_num_points;
    hipMalloc(&device_num_points,sizeof(int));
    // copy data to gpu
    hipMemcpy(device_points, host_points, num_points * 3 * sizeof(float), hipMemcpyHostToDevice);
    delete[] host_points;


    HashData hash;
    hash.allocate(true);

    HashData *d_hashdata;

    int count=0;
    int *d_count;

    hipMalloc(&d_hashdata,sizeof(HashData));
    hipMalloc(&d_count,sizeof(int));

    hipMemcpy(d_hashdata,&hash,sizeof(HashData),hipMemcpyHostToDevice);
    hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);

    dim3 blockSize(1024);
    dim3 gridSize((num_points + blockSize.x - 1) / blockSize.x);

    test<<<gridSize,blockSize>>>(d_hashdata,d_count,device_points,num_points);
    hipDeviceSynchronize();
    hash.free();
    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);    
    std::cout<<"count: "<<count<<std::endl;
    
    hipFree(d_hashdata);
    hipFree(device_points);
    
    hipFree(d_count);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after kernel launch: %s\n", hipGetErrorString(err));
    }
    return 0;
}
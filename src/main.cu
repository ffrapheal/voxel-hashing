#include "hip/hip_runtime.h"
#include<iostream>
#include"VoxelHash.h"
#include<hip/hip_runtime.h>
#include<>

__global__ void test(HashData * hash,int * d_count)
{
    float a=56*blockIdx.x;
    float b=56*blockIdx.x;
    float c=56*blockIdx.x;
    float3 worldpos=make_float3(a,b,c);
    uint pos=hash->computeHashPos(worldpos);
    return;
}
int main()
{
    HashData hash;
    HashParams params;
    hash.allocate(true);
    HashData *d_hashdata;
    int count=0;
    int *d_count;
    hipMalloc(&d_hashdata,sizeof(HashData));
    hipMalloc(&d_count,sizeof(int));
    hipMemcpy(d_hashdata,&hash,sizeof(HashData),hipMemcpyHostToDevice);
    hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
    dim3 blockSize(1,1);
    dim3 gridSize(1000);
    test<<<gridSize,blockSize>>>(d_hashdata,d_count);
    hipDeviceSynchronize();
    hash.free();
    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);    
    std::cout<<"count: "<<count<<std::endl;
    hipFree(d_hashdata);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after kernel launch: %s\n", hipGetErrorString(err));
    }
    return 0;
}
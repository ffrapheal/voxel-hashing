#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <string>
#include "VoxelHash.h"
#include <hip/hip_runtime.h>
#include <>
#include <pcl/kdtree/kdtree_flann.h>
#include <pcl/point_types.h>
#include <pcl/point_cloud.h>
#include <pcl/features/normal_3d.h>
#include <pcl/search/kdtree.h>
#include <eigen3/Eigen/src/Eigenvalues/SelfAdjointEigenSolver.h>
#include "CUDAMarchingCubesHashSDF.h"
#include <yaml-cpp/yaml.h>
#include <pcl/io/pcd_io.h>
extern __constant__ HashParams c_hashParams;

struct PointXYZINormal {
    float x, y, z;
    float intensity;
    float normal_x, normal_y, normal_z;
};

bool loadPCDFile(const std::string& filename, std::vector<PointXYZINormal>& points) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        return false;
    }

    std::string line;
    bool header = true;
    while (std::getline(file, line)) {
        if (header) {
            if (line == "DATA ascii") {
                header = false;
            }
            continue;
        }

        std::istringstream iss(line);
        PointXYZINormal point;
        if (!(iss >> point.x >> point.y >> point.z >> point.intensity >> point.normal_x >> point.normal_y >> point.normal_z)) {
            break;
        }
        points.push_back(point);
    }

    file.close();
    return true;
}

__global__ void test(HashData * hash,int * count,float * pos,int num_points)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=num_points) return;
    float3 worldpos=make_float3(pos[idx*3],pos[idx*3+1],pos[idx*3+2]);
    //assume (a b c) is the coordinate of a point.
    int3 voxelpos = hash->worldToVirtualVoxelPos(worldpos);
    uint hashpos=hash->computeHashPos(worldpos);
    hash->insertHashEntryElement(worldpos);
    __threadfence();
    HashEntry curr = hash->getHashEntryForWorldPos(worldpos);
    uint h = hash->computeHashPos(worldpos);
    if(curr.ptr!=FREE_ENTRY)
    {
        atomicAdd(&count[0], 1);
        Voxel * v = hash->getVoxel(worldpos);
        printf("voxelpos: %d %d %d\n",voxelpos.x,voxelpos.y,voxelpos.z);
        printf("voxel: %f %f %f\n",v->sdf_sum,v->weight_sum,v->sdf_sum/v->weight_sum);
    }
    return;
}

__global__ void extract_ply(HashData* hash,float3* d_voxels,float* count)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=2000000*10) return;
    if(hash->d_hash[idx].ptr!=FREE_ENTRY)
    {
        //printf("1111\n");
        for(int i=0;i<512;i++)
        {
            //printf("2222\n");
            if(hash->d_SDFBlocks[hash->d_hash[idx].ptr+i].weight_sum!=0)
            {
                uint3 uvoxellocalpos = hash->delinearizeVoxelIndex(i);
                int3 voxellocalpos = make_int3(uvoxellocalpos.x,uvoxellocalpos.y,uvoxellocalpos.z);
                int3 voxelpos = voxellocalpos+hash->d_hash[idx].pos;
                float3 worldpos = hash->virtualVoxelPosToWorld(voxelpos);
                float a = atomicAdd(count,1.0f) + 0.001;
                //printf("%f\n",*count);
                d_voxels[(int)a]=worldpos;
            }
        }
    }
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}

int main() {
    std::vector<PointXYZINormal> cloud;
    if (!loadPCDFile("/home/zzz/code/hash/point_cloud_7377_points.pcd", cloud)) {
        std::cerr << "Failed to read PCD file" << std::endl;
        return -1;
    }

    std::cout << "Point cloud size: " << cloud.size() << std::endl;
    size_t num_points = cloud.size();
    float3* host_points = new float3[num_points];
    float3* host_normals = new float3[num_points];

    // 使用更快的方法计算法线
    pcl::NormalEstimation<pcl::PointXYZ, pcl::Normal> ne;
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud_ptr(new pcl::PointCloud<pcl::PointXYZ>());
    pcl::PointCloud<pcl::Normal>::Ptr cloud_normals(new pcl::PointCloud<pcl::Normal>());
    pcl::search::KdTree<pcl::PointXYZ>::Ptr tree(new pcl::search::KdTree<pcl::PointXYZ>());

    for (size_t j = 0; j < num_points; ++j) {
        pcl::PointXYZ point;
        point.x = cloud[j].x;
        point.y = cloud[j].y;
        point.z = cloud[j].z;
        cloud_ptr->points.push_back(point);
    }

    tree->setInputCloud(cloud_ptr);
    ne.setInputCloud(cloud_ptr);
    ne.setSearchMethod(tree);
    ne.setKSearch(5);
    ne.compute(*cloud_normals);

    for (size_t i = 0; i < num_points; ++i) {
        host_points[i] = make_float3(cloud[i].x, cloud[i].y, cloud[i].z);
        host_normals[i] = make_float3(cloud_normals->points[i].normal_x, cloud_normals->points[i].normal_y, cloud_normals->points[i].normal_z);
    }
    
    // allocate memory on gpu
    float3* device_points;
    hipMalloc(&device_points, num_points * sizeof(float3));
    float3* device_normals;  
    hipMalloc(&device_normals, num_points * sizeof(float3));
    // copy data to gpu
    hipMemcpy(device_points, host_points, num_points * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(device_normals, host_normals, num_points * sizeof(float3), hipMemcpyHostToDevice);
    delete[] host_points;
    delete[] host_normals;
    // initialize hash data
    HashData hash;
    hash.allocate(true);

    HashData *d_hashdata;

    float count=0;
    float *d_count;

    hipMalloc(&d_hashdata,sizeof(HashData));
    hipMalloc(&d_count,sizeof(float));

    hipMemcpy(d_hashdata,&hash,sizeof(HashData),hipMemcpyHostToDevice);
    hipMemcpy(d_count,&count,sizeof(float),hipMemcpyHostToDevice);

    dim3 blockSize(1024);
    dim3 gridSize((num_points + blockSize.x - 1) / blockSize.x);

    // test<<<gridSize,blockSize>>>(d_hashdata,d_count,device_points,num_points);
    updatesdfframe<<<gridSize,blockSize>>>(d_hashdata,device_points,device_normals,num_points);
    hipDeviceSynchronize();

    
    float3* d_voxels;
    hipMalloc(&d_voxels, num_points * sizeof(float3));
    
    dim3 blockSize1(1024);
    dim3 gridSize1((2000000*10 + blockSize.x-1) / blockSize.x);

    extract_ply<<<gridSize1,blockSize1>>>(d_hashdata,d_voxels,d_count);
    checkCudaError(hipDeviceSynchronize());

    // 分配主机内存并从GPU复制点云数据
    hipMemcpy(&count,d_count,sizeof(float),hipMemcpyDeviceToHost);
    printf("count: %f\n",count);
    float3* host_voxels = new float3[(int)(count+0.001)];
    hipMemcpy(host_voxels, d_voxels, (int)(count+0.001) * sizeof(float3), hipMemcpyDeviceToHost);

    // 释放GPU内存
    // hipFree(d_voxels);
    // hipFree(device_points);
    // hipFree(device_normals);
    // hipFree(d_hashdata);
    // hipFree(d_count);

    // 创建PCL点云对象
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud_out(new pcl::PointCloud<pcl::PointXYZ>);
    cloud_out->width = (int)(count+0.001);
    cloud_out->height = 1;
    cloud_out->points.resize((int)(count+0.001));

    // 将float3数据转换为PCL点云格式
    for(int i = 0; i < (int)(count+0.001); i++) {
        cloud_out->points[i].x = host_voxels[i].x;
        cloud_out->points[i].y = host_voxels[i].y;
        cloud_out->points[i].z = host_voxels[i].z;
    }

    // 保存为PCD文件
    pcl::io::savePCDFileASCII("output.pcd", *cloud_out);
    delete[] host_voxels;
    hash.free();

    return 0;
    // Marching Cubes to extract mesh
    // MarchingCubesParams mcParams = CUDAMarchingCubesHashSDF::parametersFromGlobalAppState(10000000, 0, 0.05, 2000000);
    // CUDAMarchingCubesHashSDF marchingCubes(mcParams);
    // HashParams hashParams;
    // YAML::Node config = YAML::LoadFile("/home/zzz/code/hash/config/hash_params.yaml");
    // hashParams.m_hashNumBuckets = config["hashNumBuckets"].as<unsigned int>();
    // hashParams.m_hashBucketSize = config["hashBucketSize"].as<unsigned int>();
    // hashParams.m_SDFBlockSize = config["SDFBlockSize"].as<unsigned int>();
    // hashParams.m_virtualVoxelSize = config["virtualVoxelSize"].as<float>();
    // hashParams.m_maxIntegrationDistance = config["maxIntegrationDistance"].as<float>();
    // hashParams.m_truncScale = config["truncScale"].as<float>();
    // hashParams.m_truncation = config["truncation"].as<float>();
    // hashParams.m_integrationWeightSample = config["integrationWeightSample"].as<float>();
    // hashParams.m_integrationWeightMax = config["integrationWeightMax"].as<float>();
    
    // marchingCubes.extractIsoSurface(hash, hashParams, vec3f(0.0f, 0.0f, 0.0f), vec3f(1.0f, 1.0f, 1.0f), false);
    // hipDeviceSynchronize();
    // marchingCubes.export_ply("output_mesh.ply");

    // hash.free();
    // hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);    
    // std::cout<<"count: "<<count<<std::endl;
    
    // // hipFree(d_hashdata);
    // // hipFree(device_points);
    // // hipFree(device_normals);
    // // hipFree(d_count);
    // // hipError_t err = hipGetLastError();
    // // if (err != hipSuccess) {
    // //     printf("CUDA error after kernel launch: %s\n", hipGetErrorString(err));
    // // }
    // return 0;
}